//This is practice code of using cuda to do vector dot product without reduction of array
//Author: Zhaoyuan "Maxwell" Cui
//May 19, 2017

//Define certain parameters that will be used across the application
# define N 100 //The size of the vector
# define blockSize 10 
# define threadSize 10


#include <hip/hip_runtime.h>
#include<iostream>

//Prototype of the GPU kernal
__global__ void dot(float *a, float *b, float *c);

int main()
{
  //Declare variables in the host
  float *a,*b,*c,*d;
  float size_ab,size_c;
  
  //Allocate memory and initialize variables
  size_ab=sizeof(float)*N;
  size_c=sizeof(float)*blockSize;

  a=(float*)malloc(size_ab);
  b=(float*)malloc(size_ab);
  c=(float*)malloc(size_c);
  d=(float*)malloc(size_ab);

  for(int i=0;i<N;i++)
    {
      a[i]=4;
      b[i]=2;
    }

  for(int i=0;i<blockSize;i++)
    {
      c[i]=0;
      d[i]=0;
    }

  //Declare variables in the device
  float *d_a, *d_b, *d_c;

  //Allocate memory 
  hipMalloc(&d_a,size_ab);
  hipMalloc(&d_b,size_ab);
  hipMalloc(&d_c,size_c);

  //Copy the data of the host variables to the device variables
  hipMemcpy(d_a,a,size_ab,hipMemcpyHostToDevice);
  hipMemcpy(d_b,b,size_ab,hipMemcpyHostToDevice);
  hipMemcpy(d_c,c,size_c,hipMemcpyHostToDevice);

  //Launch kernal
  dot<<<blockSize,threadSize>>>(d_a,d_b,d_c);

  //Copy the result from the device variable back to the host variable
  hipMemcpy(c,d_c,size_c,hipMemcpyDeviceToHost);

  //Calculate the final result from the GPU  
  float result;
  result=0;

  for(int i=0;i<blockSize;i++)
    {
      result+=c[i];
    }

  std::cout<<"The result of the dot product from the GPU is: "<<result<<std::endl;
  
  //Do the dot product on the CPU to varify the answer
  result=0;
  for(int i=0;i<N;i++)
    {
      d[i]=a[i]*b[i];
    }

  for(int i=0;i<N;i++)
    {
      result+=d[i];
    }

  std::cout<<"The result of the dot product from the CPU is: "<<result<<std::endl;

  //Free the variables
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(a);
  free(b);
  free(c);
  free(d);

  return 0;
}

__global__ void dot(float *a, float *b, float *c)
{
  //Calculate and assign the thread id
  int tid;
  tid=blockDim.x*blockIdx.x+threadIdx.x;
  
  //Declare the shared memory inside a block
  __shared__ float cache[threadSize];
  
  //Declare a tempory variable to hold data
  float temp;

  if(tid<N)
    {
      temp=a[tid]*b[tid];
      cache[threadIdx.x]=temp;
    }

  //Wait for all the threads
  __syncthreads();
  
  //Calculate the sum of the product inside a block
  for(int i=0;i<threadSize;i++)
    {
      c[blockIdx.x]+=cache[i];
    }
}
