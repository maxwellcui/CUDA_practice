//This is practice code of using cuda to do vector dot product without reduction of array
//Author: Zhaoyuan "Maxwell" Cui
//May 30, 2017

//Define certain parameters that will be used across the application
# define N 100 //The size of the vector
# define blockSize 10 
# define threadSize 10


#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>

//Prototype of the GPU kernal
__global__ void dot(float *a, float *b, float *c);

int main()
{
  //Declare CUDA event that will evaluate the performance of the code
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Declare variables in the host
  float *a,*b,*c,*d;
  float size_ab,size_c;
  
  //Allocate memory and initialize variables
  size_ab=sizeof(float)*N;
  size_c=sizeof(float)*blockSize;

  a=(float*)malloc(size_ab);
  b=(float*)malloc(size_ab);
  c=(float*)malloc(size_c);
  d=(float*)malloc(size_ab);

  for(int i=0;i<N;i++)
    {
      a[i]=4;
      b[i]=2;
    }

  for(int i=0;i<blockSize;i++)
    {
      c[i]=0;
      d[i]=0;
    }

  //Declare variables in the device
  float *d_a, *d_b, *d_c;

  //Allocate memory 
  hipMalloc(&d_a,size_ab);
  hipMalloc(&d_b,size_ab);
  hipMalloc(&d_c,size_c);

  //Copy the data of the host variables to the device variables
  hipMemcpy(d_a,a,size_ab,hipMemcpyHostToDevice);
  hipMemcpy(d_b,b,size_ab,hipMemcpyHostToDevice);
  hipMemcpy(d_c,c,size_c,hipMemcpyHostToDevice);

  //Start event records
  hipEventRecord(start,0);

  //Launch kernal
  dot<<<blockSize,threadSize>>>(d_a,d_b,d_c);

  //Stop event records and sync
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  //Calculate the elapsed time
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,start,stop);
  
  //Copy the result from the device variable back to the host variable
  hipMemcpy(c,d_c,size_c,hipMemcpyDeviceToHost);

  //Calculate the final result from the GPU  
  float result;
  result=0;

  for(int i=0;i<blockSize;i++)
    {
      result+=c[i];
    }

  std::cout<<"The result of the dot product from the GPU is: "<<result<<std::endl;
  std::cout<<"\t-Time of calculation on GPU is:"<<elapsedTime<<"[ms]\n\n";
  
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  //Do the dot product on the CPU to varify the answer
  result=0;
  for(int i=0;i<N;i++)
    {
      d[i]=a[i]*b[i];
    }

  for(int i=0;i<N;i++)
    {
      result+=d[i];
    }

  std::cout<<"The result of the dot product from the CPU is: "<<result<<std::endl;
 
  //Free the variables
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(a);
  free(b);
  free(c);
  free(d);

  return 0;
}

__global__ void dot(float *a, float *b, float *c)
{
  //Calculate and assign the thread id
  int tid;
  tid=blockDim.x*blockIdx.x+threadIdx.x;
  
  //Declare the shared memory inside a block
  __shared__ float cache[threadSize];
  
  //Declare a tempory variable to hold data
  float temp;

  if(tid<N)
    {
      temp=a[tid]*b[tid];
      cache[threadIdx.x]=temp;
    }

  //Wait for all the threads
  __syncthreads();
  
  //Calculate the sum of the product inside a block
  for(int i=0;i<threadSize;i++)
    {
      c[blockIdx.x]+=cache[i];
    }
}
