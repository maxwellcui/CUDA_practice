//A practice code of using constant memory in dot product
//Author: Zhaoyuan "Maxwell" Cui
//Jun 13, 2017


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

#define VEC_DIM 120

//Define constant memory
__constant__ float d_vec[VEC_DIM];


//Prototype the kernal
__global__ void kernal(float* input,float* d_result);

int main()
{
  //Declare variables on host and device
  float *c_vec=new float [VEC_DIM];
  float *c_result=new float;
  float *d_result,*d_input;
  
  //Allocate memory for device variable
  hipMalloc(&d_result,sizeof(float));
  hipMalloc(&d_input,sizeof(float)*VEC_DIM);

  //Initialize host vector
  for(int i=0;i<VEC_DIM;i++)
    {
      c_vec[i]=1;
    }

  //Copy memory to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_vec),c_vec,sizeof(float)*VEC_DIM);

  //Copy memory to device input vector
  hipMemcpy(d_input,c_vec,sizeof(float)*VEC_DIM,hipMemcpyHostToDevice);

  //RUN!!
  kernal<<<1,VEC_DIM>>>(d_input,d_result);

  //Copy result from device to host
  hipMemcpy(c_result,d_result,sizeof(float),hipMemcpyDeviceToHost);

  //SYNC...
  hipDeviceSynchronize();

  //Show the result
  std::cout<<"The result is: "<<*c_result<<std::endl;

  //Free allocated memory
  delete [] c_vec;
  delete c_result;
  hipFree(d_result);
  hipFree(d_input);

  return 0;
}

__global__ void kernal(float* d_input,float* d_result)
{
  //Declare and acquire thread id
  int tid;
  tid=blockIdx.x*blockDim.x+threadIdx.x;
  
  //Declare shared memory
  __shared__ float cache[VEC_DIM];
  
  //do the calculation
  if(tid<VEC_DIM)
    {
      cache[tid]=d_input[tid]*d_vec[tid];
    }

  //Wait for all threads
  __syncthreads();

  //Initialize the return variable
  *d_result=0;

  //Caluclate the result
  for(int i=0;i<VEC_DIM;i++)
    {
      *d_result+=cache[i];
    }
}
