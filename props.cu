// This program will print out some of the properties of the GPU that is being used


#include <hip/hip_runtime.h>
#include<iostream>

int main()
{
  int deviceId;
  int warpSize;
  int computeCapabilityMajor;
  int computeCapabilityMinor;
  int multiProcessorCount;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props,deviceId);

  warpSize=props.warpSize;
  computeCapabilityMajor=props.major;
  computeCapabilityMinor=props.minor;
  multiProcessorCount=props.multiProcessorCount;

  std::cout << "The device Id is: "<<deviceId
            << "\nThe major compute capability is: "<<computeCapabilityMajor
            << "\nThe minor compute capability is: "<<computeCapabilityMinor
            << "\nThe number of SM is: "<<multiProcessorCount
            << "\nThe warp size is: "<<warpSize<<std::endl;
}
