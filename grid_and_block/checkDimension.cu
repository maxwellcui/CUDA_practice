//This is a practice code of checking the dimension and coordinate of 
//block and grid from both host and device
//Author: Maxwell Cui
//Jan 26, 2018


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void checkIndex()
{
  printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d)\nblockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n\n",threadIdx.x,threadIdx.y,threadIdx.z,blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,blockDim.z,gridDim.x,gridDim.y,gridDim.z);
}

int main()
{
  //Number of elements
  int nElem=6;
  
  dim3 block (3);
  dim3 grid ((nElem+block.x-1)/block.x);

  printf("gird.x %d, gird.x %d, gird.x %d\n", grid.x, grid.y, grid.z);
  printf("block.x %d, block.x %d, block.x %d\n", block.x, block.y, block.z);

  checkIndex<<<grid,block>>>();

  hipDeviceReset();

  return 0;
}
