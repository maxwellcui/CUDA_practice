#include "hip/hip_runtime.h"
//This is a code for the kernel basics and also the error handling
//Author: Zhaoyuan "Maxwell" Cui

#include<hip/hip_runtime.h>
#include<stdio.h>

#define CHECK(call)
{
  const hipError_t error=call;
  if(error!=hipSuccess)
    {
      printf("Error: %s:%d, ", __FILE__, __LINE__);
      printf("code:%d, reason: %s\n",error,hipGetErrorString(error));
      exit(1);
    }
}

__global__ kernel(int *A, int *B, int *C)
{
  int i;
  i=threadIdx.x;
  C[i]=A[i]+B[i];
}

int main()
{
  int nElm=100;
  int *h_A, *h_B, *h_C;
  int size=nElm*sizeof(float);
  h_A=malloc(size);
  h_B=malloc(size);
  h_C=malloc(size);

  for(int i=0;i<nElm;i++)
    {
      h_A[i]=1;
      h_B[i]=2;
      h_C[i]=0;
    }

  int *d_A,*d_B,*d_C;
  CHECK(hipMalloc(&d_A,size));
  CHECK(hipMalloc(&d_B,size));
  CHECK(hipMalloc(&d_C,size));

  CHECK(cudaMemcoy(d_A,h_A,size,hipMemcpyHostToDevice));
  CHECK(cudaMemcoy(d_B,h_B,size,hipMemcpyHostToDevice));
  CHECK(cudaMemcoy(d_C,h_C,size,hipMemcpyHostToDevice));
  
  dim3 block (10);
  dim3 grid ((nElm+block.x-1)/block.x);

  CHECK(kernel<<<grid,block>>>(d_A,d_B,d_C));
  CHECK(hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost));

  for(int i=0;i<nElm;i++)
    {
      if(h_C[i]!=3}
      {
	printf("Calculation error!\n");
	exit(1);
      }
    }
  
  return 0;
  
}
