//
//This is a code for the kernel basics and also the error handling
//Author: Zhaoyuan "Maxwell" Cui

#include<hip/hip_runtime.h>
#include<stdio.h>

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
    {\
      printf("Error: %s:%d, ", __FILE__, __LINE__);\
      printf("code:%d, reason: %s\n",error,hipGetErrorString(error));\
      exit(1);\
    }\
}\

__global__ void kernel(float *A, float *B, float *C)
{
  int i;
  i=blockIdx.x*blockDim.x+threadIdx.x;
  C[i]=A[i]+B[i];
  printf("A: %f B:%f C:%f\n",A[i],B[i],C[i]);
}

int main()
{
  int nElm=100;
  float *h_A, *h_B, *h_C;
  int size=nElm*sizeof(float);
  h_A=(float*)malloc(size);
  h_B=(float*)malloc(size);
  h_C=(float*)malloc(size);

  for(int i=0;i<nElm;i++)
    {
      h_A[i]=1;
      h_B[i]=2;
      h_C[i]=0;
    }

  float *d_A,*d_B,*d_C;
  CHECK(hipMalloc(&d_A,size));
  CHECK(hipMalloc(&d_B,size));
  CHECK(hipMalloc(&d_C,size));

  CHECK(hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_C,h_C,size,hipMemcpyHostToDevice));
  
  dim3 block (10);
  dim3 grid ((nElm+block.x-1)/block.x);

  kernel<<<grid,block>>>(d_A,d_B,d_C);
  CHECK(hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost));

  for(int i=0;i<nElm;i++)
    {
      if(h_C[i]!=3)
      {
	printf("%f.\n",h_C[i]);
	printf("Calculation error!\n");
	exit(1);
      }
    }

  free(h_A);
  free(h_B);
  free(h_C);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  
  return 0;
  
}
